#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void prefix_scan_8_0_shmem(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs[256];

    if (threadIdx.y < 8)
        sh_pfxs[threadIdx.y * 32 + threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {
        int value = keys[i];
        atomicAdd(sh_pfxs + value, 1);
    }

    __syncthreads();

    if (threadIdx.y < 8) {
        int off = threadIdx.y * 32 + threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs[off]);
    }
}



__global__
void prefix_scan_8_0_shmem_shortseg(unsigned char *keys, int *pfxs) {
    const int blksz = 256;
    const int grpsz = 8192;
    const int tid = threadIdx.y * 32 + threadIdx.x;
    __shared__ int shr_pfxs[blksz];

    shr_pfxs[tid] = 0;
    __syncthreads();

    // TODO: this introduces a hard upper limit of 512M keys (3GB) sorted in a
    // pass. It'll be a while before we get the 8GB cards needed to do this.
    int i = tid + grpsz * blockIdx.x;

    for (int j = 0; j < 32; j++) {
        int value = keys[i];
        atomicAdd(shr_pfxs + value, 1);
        i += blksz;
    }

    __syncthreads();
    pfxs[tid + blksz * blockIdx.x] = shr_pfxs[tid];
}

__global__
void crappy_split(int *pfxs, int *pfxs_out) {
    const int blksz = 256;
    const int tid = threadIdx.y * 32 + threadIdx.x;
    int i = blksz * (tid + blockIdx.x * blksz);
    int i_bound = i + blksz;
    int val = 0;
    for (; i < i_bound; i++) {
        pfxs_out[i] = val;
        val += pfxs[i];
    }
}

__global__
void prefix_sum(int *pfxs, int nitems, int *out_pfxs, int *out_sums) {
    const int blksz = 256;
    const int tid = threadIdx.y * 32 + threadIdx.x;
    int val = 0;
    for (int i = tid; i < nitems; i += blksz) val += pfxs[i];

    out_pfxs[tid] = val;

    // I know there's a better way to implement this summing network,
    // but it's not a time-critical piece of code.
    __shared__ int sh_pfxs[blksz];
    sh_pfxs[tid] = val;
    val = 0;
    __syncthreads();
    // Intentionally exclusive indexing here, val{0} should be 0
    for (int i = 0; i < tid; i++) val += sh_pfxs[i];
    out_sums[tid] = val;

    // Here we shift things over by 1, to make retrieving the
    // indices and differences easier in the sorting step.
    int i;
    for (i = tid; i < nitems; i += blksz) {
        int t = pfxs[i];
        pfxs[i] = val;
        val += t;
    }
    // Now write the last column and we're done.
    pfxs[i] = val;
}

__global__
void sort_8(unsigned char *keys, int *sorted_keys, int *pfxs) {
    const int grpsz = 8192;
    const int blksz = 256;
    const int tid = threadIdx.y * 32 + threadIdx.x;
    const int blk_offset = grpsz * blockIdx.x;
    __shared__ int shr_pfxs[blksz];

    if (threadIdx.y < 8) {
        int pfx_i = blksz * blockIdx.x + tid;
        shr_pfxs[tid] = pfxs[pfx_i];
    }
    __syncthreads();

    int i = tid;
    for (int j = 0; j < 32; j++) {
        int value = keys[i+blk_offset];
        int offset = atomicAdd(shr_pfxs + value, 1);
        sorted_keys[offset] = value;
        i += blksz;
    }
}

__global__
void sort_8_a(unsigned char *keys, int *sorted_keys, int *pfxs, int *split) {
    const int grpsz = 8192;
    const int blksz = 256;
    const int tid = threadIdx.y * 32 + threadIdx.x;
    const int blk_offset = grpsz * blockIdx.x;
    __shared__ int shr_pfxs[blksz];
    __shared__ int shr_offs[blksz];
    __shared__ int defer[grpsz];

    const int pfx_i = blksz * blockIdx.x + tid;
    shr_pfxs[tid] = pfxs[pfx_i];
    shr_offs[tid] = split[pfx_i];
    __syncthreads();

    for (int i = tid; i < grpsz; i += blksz) {
        int value = keys[i+blk_offset];
        int offset = atomicAdd(shr_offs + value, 1);
        defer[offset] = value;
    }
    //shr_pfxs[tid] = pfxs[pfx_i];
    __syncthreads();

    for (int i = tid; i < grpsz; i += blksz) {
        int value = defer[i];
        int offset = shr_pfxs[value] + i - (shr_offs[value] - shr_offs[0]);
        sorted_keys[offset] = value;
    }
}



__global__
void prefix_scan_8_0_shmem_lessconf(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs_banked[256][32];

    for (int i = threadIdx.y; i < 256; i += blockDim.y)
        sh_pfxs_banked[i][threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {
        int value = keys[i];
        atomicAdd(&(sh_pfxs_banked[value][threadIdx.x]), 1);
    }

    __syncthreads();

    for (int i = threadIdx.y; i < 256; i += blockDim.y) {
        for (int j = 16; j > 0; j = j >> 1)
            if (j > threadIdx.x)
                sh_pfxs_banked[i][threadIdx.x] += sh_pfxs_banked[i][j+threadIdx.x];
        __syncthreads();
    }

    if (threadIdx.y < 8) {
        int off = threadIdx.y * 32 + threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs_banked[off][0]);
    }

}

__global__
void prefix_scan_5_0_popc(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs[32];

    if (threadIdx.y == 0) sh_pfxs[threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    int sum = 0;

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {

        int value = keys[i];
        int test = __ballot(value & 1);
        if (!(threadIdx.x & 1)) test = ~test;

        int popc_res = __ballot(value & 2);
        if (!(threadIdx.x & 2)) popc_res = ~popc_res;
        test &= popc_res;

        popc_res = __ballot(value & 4);
        if (!(threadIdx.x & 4)) popc_res = ~popc_res;
        test &= popc_res;

        popc_res = __ballot(value & 8);
        if (!(threadIdx.x & 8)) popc_res = ~popc_res;
        test &= popc_res;

        popc_res = __ballot(value & 16);
        if (!(threadIdx.x & 16)) popc_res = ~popc_res;
        test &= popc_res;

        sum += __popc(test);
    }

    atomicAdd(sh_pfxs + threadIdx.x + 0,   sum);
    __syncthreads();

    if (threadIdx.y == 0) {
        int off = threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs[off]);
    }
}


__global__
void prefix_scan_8_0_popc(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs[256];

    if (threadIdx.y < 8)
        sh_pfxs[threadIdx.y * 32 + threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    int sum_000 = 0;
    int sum_001 = 0;
    int sum_010 = 0;
    int sum_011 = 0;
    int sum_100 = 0;
    int sum_101 = 0;
    int sum_110 = 0;
    int sum_111 = 0;

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {

        int value = keys[i];
        int test_000 = __ballot(value & 1);
        if (!(threadIdx.x & 1)) test_000 = ~test_000;

        int popc_res = __ballot(value & 2);
        if (!(threadIdx.x & 2)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 4);
        if (!(threadIdx.x & 4)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 8);
        if (!(threadIdx.x & 8)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 16);
        if (!(threadIdx.x & 16)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 32);
        int test_001 = test_000 & popc_res;
        popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 64);
        int test_010 = test_000 & popc_res;
        int test_011 = test_001 & popc_res;
        popc_res = ~popc_res;
        test_000 &= popc_res;
        test_001 &= popc_res;

        popc_res = __ballot(value & 128);
        int test_100 = test_000 & popc_res;
        int test_101 = test_001 & popc_res;
        int test_110 = test_010 & popc_res;
        int test_111 = test_011 & popc_res;
        popc_res = ~popc_res;
        test_000 &= popc_res;
        test_001 &= popc_res;
        test_010 &= popc_res;
        test_011 &= popc_res;

        sum_000 += __popc(test_000);
        sum_001 += __popc(test_001);
        sum_010 += __popc(test_010);
        sum_011 += __popc(test_011);
        sum_100 += __popc(test_100);
        sum_101 += __popc(test_101);
        sum_110 += __popc(test_110);
        sum_111 += __popc(test_111);
    }

    atomicAdd(sh_pfxs + (threadIdx.x + 0),   sum_000);
    atomicAdd(sh_pfxs + (threadIdx.x + 32),  sum_001);
    atomicAdd(sh_pfxs + (threadIdx.x + 64),  sum_010);
    atomicAdd(sh_pfxs + (threadIdx.x + 96),  sum_011);
    atomicAdd(sh_pfxs + (threadIdx.x + 128), sum_100);
    atomicAdd(sh_pfxs + (threadIdx.x + 160), sum_101);
    atomicAdd(sh_pfxs + (threadIdx.x + 192), sum_110);
    atomicAdd(sh_pfxs + (threadIdx.x + 224), sum_111);

    __syncthreads();

    if (threadIdx.y < 8) {
        int off = threadIdx.y * 32 + threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs[off]);
    }
}

