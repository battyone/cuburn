#include <hip/hip_runtime.h>
#include <stdio.h>

#define s(x) #x

__global__
void prefix_scan_8_0_shmem(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs[256];

    if (threadIdx.y < 8)
        sh_pfxs[threadIdx.y * 32 + threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {
        int value = keys[i];
        atomicAdd(sh_pfxs + value, 1);
    }

    __syncthreads();

    if (threadIdx.y < 8) {
        int off = threadIdx.y * 32 + threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs[off]);
    }
}

#define GRP_RDX_FACTOR (GRPSZ / RDXSZ)
#define GRP_BLK_FACTOR (GRPSZ / BLKSZ)
#define GRPSZ 8192
#define RDXSZ 256
#define BLKSZ 512

__global__
void prefix_scan_8_0(unsigned short *offsets, int *pfxs,
                     const unsigned short *keys) {
    const int tid = threadIdx.x;
    __shared__ int shr_pfxs[RDXSZ];

    if (tid < RDXSZ) shr_pfxs[tid] = 0;
    __syncthreads();
    int i = tid + GRPSZ * blockIdx.x;

    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        // TODO: compiler smart enough to turn this into a BFE?
        // TODO: should this just be two functions with fixed shifts?
        // TODO: separate or integrated loop vars? unrolling?
        int value = keys[i] & 0xff;
        offsets[i] = atomicAdd(shr_pfxs + value, 1);
        i += BLKSZ;
    }

    __syncthreads();
    if (tid < RDXSZ) pfxs[tid + RDXSZ * blockIdx.x] = shr_pfxs[tid];
}

__global__
void prefix_scan_8_8(unsigned short *offsets, int *pfxs,
                     const unsigned short *keys) {
    const int tid = threadIdx.x;
    const int blk_offset = GRPSZ * blockIdx.x;
    __shared__ int shr_pfxs[RDXSZ];
    __shared__ int shr_lo_radix;
    __shared__ int shr_rerun;

    if (tid < RDXSZ) {
        shr_pfxs[tid] = 0;
        if (tid == 0) {
            shr_lo_radix = keys[GRPSZ * blockIdx.x] & 0xff;
            shr_rerun = 0;
        }
    }
    __syncthreads();

    int ran = 0;

    int i = tid;
    while (i < GRPSZ) {
        int value = keys[i + blk_offset];
        int lo_radix = value & 0xff;
        if (shr_lo_radix < lo_radix) {
            shr_rerun = 1;
        } else if (shr_lo_radix == lo_radix) {
            int radix = (value >> 8) & 0xff;
            offsets[i + blk_offset] = atomicAdd(shr_pfxs + radix, 1);
            ran = 1;
        } else if (shr_lo_radix > lo_radix && !ran) {
            // For reasons I have yet to bother assessing, the optimizer
            // mangles this function unless it also includes code that runs on
            // this case. This code should never actually run, though. In
            // fact, 'ran' could be eliminated entirely, but for this.
            offsets[i] = offsets[i];
        }

        __syncthreads();
        if (shr_rerun) {
            if (tid == 0) {
                shr_lo_radix += 1;
                shr_rerun = 0;
            }
            __syncthreads();
        } else {
            i += blockDim.x;
            ran = 0;
        }
    }

    __syncthreads();
    if (tid < RDXSZ) pfxs[tid + RDXSZ * blockIdx.x] = shr_pfxs[tid];
}

__global__
void prefix_scan_8_0_shmem_shortseg(unsigned char *keys, int *pfxs) {
    const int tid = threadIdx.y * 32 + threadIdx.x;
    __shared__ int shr_pfxs[RDXSZ];

    if (tid < RDXSZ) shr_pfxs[tid] = 0;
    __syncthreads();

    // TODO: this introduces a hard upper limit of 512M keys (3GB) sorted in a
    // pass. It'll be a while before we get the 8GB cards needed to do this.
    int i = tid + GRPSZ * blockIdx.x;

    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        int value = keys[i];
        atomicAdd(shr_pfxs + value, 1);
        i += BLKSZ;
    }

    __syncthreads();
    if (tid < RDXSZ) pfxs[tid + RDXSZ * blockIdx.x] = shr_pfxs[tid];
}

__global__
void crappy_split(int *pfxs, int *pfxs_out) {
    const int blksz = 256;
    const int tid = threadIdx.y * 32 + threadIdx.x;
    int i = blksz * (tid + blockIdx.x * blksz);
    int i_bound = i + blksz;
    int val = 0;
    for (; i < i_bound; i++) {
        pfxs_out[i] = val;
        val += pfxs[i];
    }
}

__global__
void better_split(int *pfxs_out, const int *pfxs) {
    // This one must be launched as 32x1, regardless of BLKSZ.
    const int tid = threadIdx.x;
    const int tid5 = tid << 5;
    __shared__ int swap[1024];

    int base = RDXSZ * 32 * blockIdx.x;

    int value = 0;

    // Performs a fast "split" (don't know why I called it that, will rename
    // soon). For each entry in pfxs (corresponding to the number of elements
    // per radix in a group), this writes the exclusive prefix sum for that
    // group. This is in fact a bunch of serial prefix sums in parallel, and
    // not a parallel prefix sum.
    //
    // The contents of 32 group radix counts are loaded in 32-element chunks
    // into shared memory, rotated by 1 unit each group to avoid bank
    // conflicts. Each thread in the warp sums across each group serially,
    // updating the values as it goes, then the results are written coherently
    // to global memory.
    //
    // This leaves the SM underloaded, as this only allows 12 warps per SM. It
    // might be better to halve the chunk size and lose some coalescing
    // efficiency; need to benchmark. It's a relatively cheap step, though.

    for (int j = 0; j < 8; j++) {
        int jj = j << 5;
        for (int i = 0; i < 32; i++) {
            int base_offset = (i << 8) + jj + base + tid;
            int swap_offset = (i << 5) + ((i + tid) & 0x1f);
            swap[swap_offset] = pfxs[base_offset];
        }

#pragma unroll
        for (int i = 0; i < 32; i++) {
            int swap_offset = tid5 + ((i + tid) & 0x1f);
            int tmp = swap[swap_offset];
            swap[swap_offset] = value;
            value += tmp;
        }

        for (int i = 0; i < 32; i++) {
            int base_offset = (i << 8) + jj + base + tid;
            int swap_offset = (i << 5) + ((i + tid) & 0x1f);
            pfxs_out[base_offset] = swap[swap_offset];
        }
    }
}


__global__
void prefix_sum(int *pfxs, const int nitems) {
    // Needs optimizing (later). Should be rolled into split.
    // Must launch 256 threads.
    const int tid = threadIdx.x;
    const int blksz = 256;
    int val = 0;
    for (int i = tid; i < nitems; i += blksz) val += pfxs[i];

    // I know there's a better way to implement this summing network,
    // but it's not a time-critical piece of code.
    __shared__ int sh_pfxs[blksz];
    sh_pfxs[tid] = val;
    val = 0;
    __syncthreads();
    // Intentionally exclusive indexing here, val{0} should be 0
    for (int i = 0; i < tid; i++) val += sh_pfxs[i];

    int i;
    for (i = tid; i < nitems; i += blksz) {
        int t = pfxs[i];
        pfxs[i] = val;
        val += t;
    }
}

__global__
void sort_8(unsigned char *keys, int *sorted_keys, int *pfxs) {
    const int tid = threadIdx.x;
    const int blk_offset = GRPSZ * blockIdx.x;
    __shared__ int shr_pfxs[RDXSZ];

    if (tid < RDXSZ) shr_pfxs[tid] = pfxs[RDXSZ * blockIdx.x + tid];
    __syncthreads();

    int i = tid;
    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        int value = keys[i+blk_offset];
        int offset = atomicAdd(shr_pfxs + value, 1);
        sorted_keys[offset] = value;
        i += BLKSZ;
    }
}


#undef BLKSZ
#define BLKSZ 1024
__global__
void sort_8_a(unsigned char *keys, int *sorted_keys,
              const int *pfxs, const int *split) {
    const int tid = threadIdx.x;
    const int blk_offset = GRPSZ * blockIdx.x;
    __shared__ int shr_offs[RDXSZ];
    __shared__ int defer[GRPSZ];

    const int pfx_i = RDXSZ * blockIdx.x + tid;
    if (tid < RDXSZ) shr_offs[tid] = split[pfx_i];
    __syncthreads();

    for (int i = tid; i < GRPSZ; i += BLKSZ) {
        int value = keys[i+blk_offset];
        int offset = atomicAdd(shr_offs + value, 1);
        defer[offset] = value;
    }
    __syncthreads();

    // This calculation is a bit odd.
    //
    // For a given radix value 'r', shr_offs[r] currently holds the first index
    // of the *next* radix in defer[] (i.e.  if there are 28 '0'-radix values
    // in defer[], shr_offs[0]==28). We want to get back to a normal exclusive
    // prefix, so we subtract shr_offs[0] from everything.
    //
    // In the next block, we want to be able to find the correct position for a
    // value in defer[], given that value's index 'i' and its radix 'r'. This
    // requires two values: the destination index in sorted_keys[] of the first
    // value in the group with radix 'r' (given by pfxs[BASE + r]), and the
    // number of radix-'r' values before this one in defer[]. So, ultimately,
    // we want an equation in the inner loop below that looks like this:
    //
    //      int dst_offset = pfxs[r] + i - (shr_offs[r] - shr_offs[0]);
    //      sorted_keys[dst_offset] = defer[i];
    //
    // Of course, this generates tons of memory lookups and bank conflicts so
    // we precombine some of this here.
    int off0 = shr_offs[0];
    if (tid < RDXSZ) shr_offs[tid] = pfxs[0] - (shr_offs[tid] - off0);
    __syncthreads();

    int i = tid;
#pragma unroll
    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        int value = defer[i];
        int offset = shr_offs[value] + i;
        sorted_keys[offset] = value;
        i += BLKSZ;
    }
}

__global__
void convert_offsets(
        unsigned short *offsets,    // input and output
        const int *split,
        const unsigned short *keys,
        const int shift
    ) {
    const int tid = threadIdx.x;
    const int blk_offset = GRPSZ * blockIdx.x;
    const int rdx_offset = RDXSZ * blockIdx.x;
    __shared__ int shr_offsets[GRPSZ];
    __shared__ int shr_split[RDXSZ];

    if (tid < RDXSZ) shr_split[tid] = split[rdx_offset + tid];
    __syncthreads();

    for (int i = tid; i < GRPSZ; i += BLKSZ) {
        int r = (keys[blk_offset + i] >> shift) & 0xff;
        int o = shr_split[r] + offsets[blk_offset + i];
        if (o < GRPSZ)
            shr_offsets[o] = i;
        else
            printf("\nWTF b:%4x i:%4x r:%2x o:%4x s:%4x og:%4x",
                    blockIdx.x, i, r, o, shr_split[r], offsets[blk_offset+i]);
    }
    __syncthreads();

    for (int i = tid; i < GRPSZ; i += BLKSZ)
        offsets[blk_offset + i] = shr_offsets[i];
}

__global__
void radix_sort_maybe(
        unsigned short *sorted_keys,
        int *sorted_values,
        const unsigned short *keys,
        const unsigned int *values,
        const unsigned short *offsets,
        const int *pfxs,
        const int *split,
        const int shift
    ) {
    const int tid = threadIdx.x;
    const int blk_offset = GRPSZ * blockIdx.x;
    const int rdx_offset = RDXSZ * blockIdx.x;
    __shared__ int shr_offs[RDXSZ];

    if (tid < RDXSZ)
        shr_offs[tid] = pfxs[rdx_offset + tid] - split[rdx_offset + tid];
    __syncthreads();

    int i = tid;
    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        int offset = offsets[blk_offset + i];
        int key = keys[blk_offset + offset];
        int radix = (key >> shift) & 0xff;
        int glob_offset = shr_offs[radix] + i;
        /*if (sorted_values[glob_offset] != 0xffffffff)
            printf("\nbad offset pos:%6x off:%4x gloff:%6x key:%4x "
                   "okey:%4x val:%8x oval:%8x",
                    i+blk_offset, offset, glob_offset, key,
                    sorted_keys[glob_offset], sorted_values[glob_offset]);*/
        sorted_keys[glob_offset] = key;
        sorted_values[glob_offset] = values[blk_offset + offset];
        i += BLKSZ;
    }
}

__global__
void radix_sort(unsigned short *sorted_keys, int *sorted_values,
                const unsigned short *keys, const unsigned int *values,
                const int *pfxs, const int *offsets, const int *split,
                const int shift) {
    const int tid = threadIdx.x;
    const int blk_offset = GRPSZ * blockIdx.x;
    __shared__ int shr_offs[RDXSZ];
    __shared__ int defer[GRPSZ];
    __shared__ unsigned char radishes[GRPSZ];

    const int pfx_i = RDXSZ * blockIdx.x + tid;
    if (tid < RDXSZ) shr_offs[tid] = split[pfx_i];
    __syncthreads();

    for (int i = tid; i < GRPSZ; i += BLKSZ) {
        int idx = i + blk_offset;
        int value = keys[idx];
        int radix = radishes[i] = (value >> shift) & 0xff;
        int offset = offsets[idx] + split[radix];
        defer[offset] = value;
    }
    __syncthreads();

    if (tid < RDXSZ) shr_offs[tid] = pfxs[tid] - shr_offs[tid];
    __syncthreads();

    // Faster to reload these or to recompute them in shmem? Need to see if we
    // can safely stash both

    int i = tid;
#pragma unroll
    for (int j = 0; j < GRP_BLK_FACTOR; j++) {
        int value = defer[i];
        int offset = shr_offs[value] + i;
        sorted_keys[offset] = value;
        i += BLKSZ;
    }
}


__global__
void prefix_scan_8_0_shmem_lessconf(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs_banked[256][32];

    for (int i = threadIdx.y; i < 256; i += blockDim.y)
        sh_pfxs_banked[i][threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {
        int value = keys[i];
        atomicAdd(&(sh_pfxs_banked[value][threadIdx.x]), 1);
    }

    __syncthreads();

    for (int i = threadIdx.y; i < 256; i += blockDim.y) {
        for (int j = 16; j > 0; j = j >> 1)
            if (j > threadIdx.x)
                sh_pfxs_banked[i][threadIdx.x] += sh_pfxs_banked[i][j+threadIdx.x];
        __syncthreads();
    }

    if (threadIdx.y < 8) {
        int off = threadIdx.y * 32 + threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs_banked[off][0]);
    }

}

__global__
void prefix_scan_5_0_popc(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs[32];

    if (threadIdx.y == 0) sh_pfxs[threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    int sum = 0;

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {

        int value = keys[i];
        int test = __ballot(value & 1);
        if (!(threadIdx.x & 1)) test = ~test;

        int popc_res = __ballot(value & 2);
        if (!(threadIdx.x & 2)) popc_res = ~popc_res;
        test &= popc_res;

        popc_res = __ballot(value & 4);
        if (!(threadIdx.x & 4)) popc_res = ~popc_res;
        test &= popc_res;

        popc_res = __ballot(value & 8);
        if (!(threadIdx.x & 8)) popc_res = ~popc_res;
        test &= popc_res;

        popc_res = __ballot(value & 16);
        if (!(threadIdx.x & 16)) popc_res = ~popc_res;
        test &= popc_res;

        sum += __popc(test);
    }

    atomicAdd(sh_pfxs + threadIdx.x + 0,   sum);
    __syncthreads();

    if (threadIdx.y == 0) {
        int off = threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs[off]);
    }
}


__global__
void prefix_scan_8_0_popc(unsigned char *keys, int nitems, int *pfxs) {
    __shared__ int sh_pfxs[256];

    if (threadIdx.y < 8)
        sh_pfxs[threadIdx.y * 32 + threadIdx.x] = 0;

    __syncthreads();

    int blksz = blockDim.x * blockDim.y;
    int cap = nitems * (blockIdx.x + 1);

    int sum_000 = 0;
    int sum_001 = 0;
    int sum_010 = 0;
    int sum_011 = 0;
    int sum_100 = 0;
    int sum_101 = 0;
    int sum_110 = 0;
    int sum_111 = 0;

    for (int i = threadIdx.y * 32 + threadIdx.x + nitems * blockIdx.x;
         i < cap; i += blksz) {

        int value = keys[i];
        int test_000 = __ballot(value & 1);
        if (!(threadIdx.x & 1)) test_000 = ~test_000;

        int popc_res = __ballot(value & 2);
        if (!(threadIdx.x & 2)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 4);
        if (!(threadIdx.x & 4)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 8);
        if (!(threadIdx.x & 8)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 16);
        if (!(threadIdx.x & 16)) popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 32);
        int test_001 = test_000 & popc_res;
        popc_res = ~popc_res;
        test_000 &= popc_res;

        popc_res = __ballot(value & 64);
        int test_010 = test_000 & popc_res;
        int test_011 = test_001 & popc_res;
        popc_res = ~popc_res;
        test_000 &= popc_res;
        test_001 &= popc_res;

        popc_res = __ballot(value & 128);
        int test_100 = test_000 & popc_res;
        int test_101 = test_001 & popc_res;
        int test_110 = test_010 & popc_res;
        int test_111 = test_011 & popc_res;
        popc_res = ~popc_res;
        test_000 &= popc_res;
        test_001 &= popc_res;
        test_010 &= popc_res;
        test_011 &= popc_res;

        sum_000 += __popc(test_000);
        sum_001 += __popc(test_001);
        sum_010 += __popc(test_010);
        sum_011 += __popc(test_011);
        sum_100 += __popc(test_100);
        sum_101 += __popc(test_101);
        sum_110 += __popc(test_110);
        sum_111 += __popc(test_111);
    }

    atomicAdd(sh_pfxs + (threadIdx.x + 0),   sum_000);
    atomicAdd(sh_pfxs + (threadIdx.x + 32),  sum_001);
    atomicAdd(sh_pfxs + (threadIdx.x + 64),  sum_010);
    atomicAdd(sh_pfxs + (threadIdx.x + 96),  sum_011);
    atomicAdd(sh_pfxs + (threadIdx.x + 128), sum_100);
    atomicAdd(sh_pfxs + (threadIdx.x + 160), sum_101);
    atomicAdd(sh_pfxs + (threadIdx.x + 192), sum_110);
    atomicAdd(sh_pfxs + (threadIdx.x + 224), sum_111);

    __syncthreads();

    if (threadIdx.y < 8) {
        int off = threadIdx.y * 32 + threadIdx.x;
        atomicAdd(pfxs + off, sh_pfxs[off]);
    }
}

